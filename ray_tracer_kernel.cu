// ray_tracer_kernel.cu
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline __device__ void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) assert(0);
   }
}



#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
// Vector operations
__device__ float3 vector_add(float3 a, float3 b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 vector_subtract(float3 a, float3 b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 vector_multiply(float3 a, float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float3 vector_multiply_vec(float3 a, float3 b) {
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

__device__ float vector_dot(float3 a, float3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 vector_cross(float3 a, float3 b) {
    return make_float3(a.y * b.z - a.z * b.y,
                       a.z * b.x - a.x * b.z,
                       a.x * b.y - a.y * b.x);
}

__device__ float3 vector_normalize(float3 v) {
    float length = sqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    if (length > 0) {
        return make_float3(v.x / length, v.y / length, v.z / length);
    }
    return v;
}

struct Ray {
    float3 origin;
    float3 direction;
};

struct Material {
    float3 color;
    float specular;
    float reflection;
    float refraction;
    float refractive_index;
};

struct Sphere {
    float3 center;
    float radius;
    Material material;
};

struct Cylinder {
    float3 center;
    float radius;
    float height;
    Material material;
};

struct Plane {
    float3 point;
    float3 normal;
    Material material;
};

struct Rectangle {
    float3 corner;
    float3 u;
    float3 v;
    Material material;
};

struct Cube {
    float3 min_point;
    float3 max_point;
    Material material;
};


__device__ bool intersect_sphere(Ray ray, Sphere sphere, float* t) {
    float3 oc = vector_subtract(ray.origin, sphere.center);
    float a = vector_dot(ray.direction, ray.direction);
    float b = 2.0f * vector_dot(oc, ray.direction);
    float c = vector_dot(oc, oc) - sphere.radius * sphere.radius;
    float discriminant = b * b - 4 * a * c;

    if (discriminant > 0) {
        float temp = (-b - sqrtf(discriminant)) / (2.0f * a);
        if (temp > 0.001f) {
            *t = temp;
            return true;
        }
        temp = (-b + sqrtf(discriminant)) / (2.0f * a);
        if (temp > 0.001f) {
            *t = temp;
            return true;
        }
    }
    return false;
}

__device__ bool intersect_cylinder(Ray ray, Cylinder cylinder, float* t) {
    // Simplified cylinder intersection (infinite height)
    float3 ro = vector_subtract(ray.origin, cylinder.center);
    float a = ray.direction.x * ray.direction.x + ray.direction.z * ray.direction.z;
    float b = 2 * (ro.x * ray.direction.x + ro.z * ray.direction.z);
    float c = ro.x * ro.x + ro.z * ro.z - cylinder.radius * cylinder.radius;

    float discriminant = b * b - 4 * a * c;
    if (discriminant < 0) return false;

    float t0 = (-b - sqrtf(discriminant)) / (2 * a);
    float t1 = (-b + sqrtf(discriminant)) / (2 * a);

    if (t0 > t1) {
        float temp = t0;
        t0 = t1;
        t1 = temp;
    }

    float y0 = ro.y + t0 * ray.direction.y;
    float y1 = ro.y + t1 * ray.direction.y;

    float cylinder_y_min = -cylinder.height / 2;
    float cylinder_y_max = cylinder.height / 2;

    if (y0 < cylinder_y_min) {
        if (y1 < cylinder_y_min) return false;
        float th = t0 + (t1 - t0) * (cylinder_y_min - y0) / (y1 - y0);
        if (th > 0) {
            *t = th;
            return true;
        }
    } else if (y0 >= cylinder_y_min && y0 <= cylinder_y_max) {
        if (t0 > 0) {
            *t = t0;
            return true;
        }
    }

    return false;
}

__device__ bool intersect_plane(Ray ray, Plane plane, float* t) {
    float denom = vector_dot(plane.normal, ray.direction);
    if (fabsf(denom) > 1e-6) {
        float3 p0l0 = vector_subtract(plane.point, ray.origin);
        *t = vector_dot(p0l0, plane.normal) / denom;
        return (*t >= 0);
    }
    return false;
}

__device__ bool intersect_rectangle(Ray ray, Rectangle rect, float* t) {
    float3 n = vector_normalize(vector_cross(rect.u, rect.v));
    float denom = vector_dot(n, ray.direction);

    if (fabsf(denom) < 1e-6) return false;

    float3 p0r0 = vector_subtract(rect.corner, ray.origin);
    *t = vector_dot(p0r0, n) / denom;

    if (*t < 0) return false;

    float3 p = vector_add(ray.origin, vector_multiply(ray.direction, *t));
    float3 vi = vector_subtract(p, rect.corner);

    float a1 = vector_dot(vi, rect.u);
    if (a1 < 0 || a1 > vector_dot(rect.u, rect.u)) return false;

    float a2 = vector_dot(vi, rect.v);
    if (a2 < 0 || a2 > vector_dot(rect.v, rect.v)) return false;

    return true;
}

__device__ bool intersect_cube(Ray ray, Cube cube, float* t) {
    float3 inv_dir = make_float3(1.0f / ray.direction.x, 1.0f / ray.direction.y, 1.0f / ray.direction.z);
    float3 t_min = vector_multiply_vec(vector_subtract(cube.min_point, ray.origin), inv_dir);
    float3 t_max = vector_multiply_vec(vector_subtract(cube.max_point, ray.origin), inv_dir);

    float3 t_near = make_float3(fminf(t_min.x, t_max.x), fminf(t_min.y, t_max.y), fminf(t_min.z, t_max.z));
    float3 t_far = make_float3(fmaxf(t_min.x, t_max.x), fmaxf(t_min.y, t_max.y), fmaxf(t_min.z, t_max.z));

    float t_near_max = fmaxf(fmaxf(t_near.x, t_near.y), t_near.z);
    float t_far_min = fminf(fminf(t_far.x, t_far.y), t_far.z);

    if (t_near_max > t_far_min || t_far_min < 0) return false;

    *t = t_near_max;
    return true;
}

__device__ float3 trace_ray(Ray ray, Sphere* spheres, int num_spheres,
                            Cylinder* cylinders, int num_cylinders,
                            Plane* planes, int num_planes,
                            Rectangle* rectangles, int num_rectangles,
                            Cube* cubes, int num_cubes,
                            float3 light_pos, int depth) {
    if (depth > 5) return make_float3(0.0f, 0.0f, 0.0f);

    float closest_t = HIP_INF_F;
    Material* closest_material = NULL;
    float3 normal;
    float3 hit_point;

    // Check sphere intersections
    for (int i = 0; i < num_spheres; i++) {
        float t;
        if (intersect_sphere(ray, spheres[i], &t) && t < closest_t) {
            closest_t = t;
            closest_material = &spheres[i].material;
            hit_point = vector_add(ray.origin, vector_multiply(ray.direction, t));
            normal = vector_normalize(vector_subtract(hit_point, spheres[i].center));
        }
    }

    // Check cylinder intersections
    for (int i = 0; i < num_cylinders; i++) {
        float t;
        if (intersect_cylinder(ray, cylinders[i], &t) && t < closest_t) {
            closest_t = t;
            closest_material = &cylinders[i].material;
            hit_point = vector_add(ray.origin, vector_multiply(ray.direction, t));
            float3 cp = vector_subtract(hit_point, cylinders[i].center);
            cp.y = 0;
            normal = vector_normalize(cp);
        }
    }

    // Check plane intersections
    for (int i = 0; i < num_planes; i++) {
        float t;
        if (intersect_plane(ray, planes[i], &t) && t < closest_t) {
            closest_t = t;
            closest_material = &planes[i].material;
            hit_point = vector_add(ray.origin, vector_multiply(ray.direction, t));
            normal = planes[i].normal;
        }
    }

    // Check rectangle intersections
    for (int i = 0; i < num_rectangles; i++) {
        float t;
        if (intersect_rectangle(ray, rectangles[i], &t) && t < closest_t) {
            closest_t = t;
            closest_material = &rectangles[i].material;
            hit_point = vector_add(ray.origin, vector_multiply(ray.direction, t));
            normal = vector_normalize(vector_cross(rectangles[i].u, rectangles[i].v));
        }
    }

    // Check cube intersections
    for (int i = 0; i < num_cubes; i++) {
        float t;
        if (intersect_cube(ray, cubes[i], &t) && t < closest_t) {
            closest_t = t;
            closest_material = &cubes[i].material;
            hit_point = vector_add(ray.origin, vector_multiply(ray.direction, t));
            float3 center = vector_multiply(vector_add(cubes[i].min_point, cubes[i].max_point), 0.5f);
            normal = vector_normalize(vector_subtract(hit_point, center));
        }
    }

    if (closest_material == NULL) {
        // Sky color
        float t = 0.5f * (ray.direction.y + 1.0f);
        return vector_add(
            vector_multiply(make_float3(1.0f, 1.0f, 1.0f), 1.0f - t),
            vector_multiply(make_float3(0.5f, 0.7f, 1.0f), t)
        );
    }

    // Compute lighting
    float3 light_dir = vector_normalize(vector_subtract(light_pos, hit_point));
    float diffuse = fmaxf(vector_dot(normal, light_dir), 0.0f);

    float3 view_dir = vector_normalize(vector_multiply(ray.direction, -1.0f));
    float3 reflect_dir = vector_normalize(vector_subtract(vector_multiply(normal, 2.0f * vector_dot(normal, light_dir)), light_dir));
    float specular = powf(fmaxf(vector_dot(view_dir, reflect_dir), 0.0f), 50.0f) * closest_material->specular;

    float3 color = vector_multiply(closest_material->color, diffuse + 0.1f);  // 0.1 for ambient light
    color = vector_add(color, make_float3(specular, specular, specular));

    // Compute reflection
    if (closest_material->reflection > 0 && depth < 5) {
        float3 reflect_dir = vector_normalize(vector_subtract(ray.direction, vector_multiply(normal, 2.0f * vector_dot(ray.direction, normal))));
        Ray reflect_ray = {hit_point, reflect_dir};
        float3 reflection = trace_ray(reflect_ray, spheres, num_spheres, cylinders, num_cylinders, planes, num_planes, rectangles, num_rectangles, cubes, num_cubes, light_pos, depth + 1);
        color = vector_add(
            vector_multiply(color, 1.0f - closest_material->reflection),
            vector_multiply(reflection, closest_material->reflection)
        );
    }

    return color;
}

extern "C"
__global__ void ray_trace_kernel(float* output, int width, int height, int samples,
                                 float* spheres_data, int num_spheres,
                                 float* cylinders_data, int num_cylinders,
                                 float* planes_data, int num_planes,
                                 float* rectangles_data, int num_rectangles,
                                 float* cubes_data, int num_cubes) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;


    hiprandState state;
    hiprand_init(y * width + x, 0, 0, &state);

    Sphere spheres[10];
    for (int i = 0; i < num_spheres; i++) {
        spheres[i].center = make_float3(spheres_data[i*11], spheres_data[i*11+1], spheres_data[i*11+2]);
        spheres[i].radius = spheres_data[i*11+3];
        spheres[i].material.color = make_float3(spheres_data[i*11+4], spheres_data[i*11+5], spheres_data[i*11+6]);
        spheres[i].material.specular = spheres_data[i*11+7];
        spheres[i].material.reflection = spheres_data[i*11+8];
        spheres[i].material.refraction = spheres_data[i*11+9];
        spheres[i].material.refractive_index = spheres_data[i*11+10];
    }

    Cylinder cylinders[10];
    for (int i = 0; i < num_cylinders; i++) {
        cylinders[i].center = make_float3(cylinders_data[i*12], cylinders_data[i*12+1], cylinders_data[i*12+2]);
        cylinders[i].radius = cylinders_data[i*12+3];
        cylinders[i].height = cylinders_data[i*12+4];
        cylinders[i].material.color = make_float3(cylinders_data[i*12+5], cylinders_data[i*12+6], cylinders_data[i*12+7]);
        cylinders[i].material.specular = cylinders_data[i*12+8];
        cylinders[i].material.reflection = cylinders_data[i*12+9];
        cylinders[i].material.refraction = cylinders_data[i*12+10];
        cylinders[i].material.refractive_index = cylinders_data[i*12+11];
    }

    Plane planes[10];
    for (int i = 0; i < num_planes; i++) {
        planes[i].point = make_float3(planes_data[i*11], planes_data[i*11+1], planes_data[i*11+2]);
        planes[i].normal = make_float3(planes_data[i*11+3], planes_data[i*11+4], planes_data[i*11+5]);
        planes[i].material.color = make_float3(planes_data[i*11+6], planes_data[i*11+7], planes_data[i*11+8]);
        planes[i].material.specular = planes_data[i*11+9];
        planes[i].material.reflection = planes_data[i*11+10];
    }

    Rectangle rectangles[10];
    for (int i = 0; i < num_rectangles; i++) {
        rectangles[i].corner = make_float3(rectangles_data[i*14], rectangles_data[i*14+1], rectangles_data[i*14+2]);
        rectangles[i].u = make_float3(rectangles_data[i*14+3], rectangles_data[i*14+4], rectangles_data[i*14+5]);
        rectangles[i].v = make_float3(rectangles_data[i*14+6], rectangles_data[i*14+7], rectangles_data[i*14+8]);
        rectangles[i].material.color = make_float3(rectangles_data[i*14+9], rectangles_data[i*14+10], rectangles_data[i*14+11]);
        rectangles[i].material.specular = rectangles_data[i*14+12];
        rectangles[i].material.reflection = rectangles_data[i*14+13];
    }

    Cube cubes[10];
    for (int i = 0; i < num_cubes; i++) {
        cubes[i].min_point = make_float3(cubes_data[i*13], cubes_data[i*13+1], cubes_data[i*13+2]);
        cubes[i].max_point = make_float3(cubes_data[i*13+3], cubes_data[i*13+4], cubes_data[i*13+5]);
        cubes[i].material.color = make_float3(cubes_data[i*13+6], cubes_data[i*13+7], cubes_data[i*13+8]);
        cubes[i].material.specular = cubes_data[i*13+9];
        cubes[i].material.reflection = cubes_data[i*13+10];
        cubes[i].material.refraction = cubes_data[i*13+11];
        cubes[i].material.refractive_index = cubes_data[i*13+12];
    }

    float3 camera_pos = make_float3(0.0f, 0.0f, 0.0f);
    float3 light_pos = make_float3(5.0f, 5.0f, 5.0f);

    float3 color = make_float3(0.0f, 0.0f, 0.0f);
    for (int s = 0; s < samples; s++) {
        float u = (float(x) + hiprand_uniform(&state)) / float(width);
        float v = (float(y) + hiprand_uniform(&state)) / float(height);

        float3 direction = vector_normalize(make_float3(
            (2.0f * u - 1.0f) * float(width) / float(height),
            -(2.0f * v - 1.0f),
            -1.0f
        ));

        Ray ray = {camera_pos, direction};
        color = vector_add(color, trace_ray(ray, spheres, num_spheres, cylinders, num_cylinders, planes, num_planes, rectangles, num_rectangles, cubes, num_cubes, light_pos, 0));
        gpuErrchk(hipPeekAtLastError());
    }

    color = vector_multiply(color, 1.0f / float(samples));

    int idx = (y * width + x) * 3;
    gpuErrchk(hipPeekAtLastError());
    output[idx] = fminf(color.x, 1.0f);
    output[idx + 1] = fminf(color.y, 1.0f);
    output[idx + 2] = fminf(color.z, 1.0f);
    gpuErrchk(hipPeekAtLastError());
}

